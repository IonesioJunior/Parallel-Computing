
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void square_1d_vector(float * d_out , float * d_in)
{
	int idx = threadIdx.x;
	d_out[idx] = d_in[idx] * d_in[idx];
}


void call_1d_parallel_computing(void)
{
	const int ARRAY_SIZE = 32;
	const int ARRAY_BYTES = ARRAY_SIZE * sizeof(float);
	
	//Host mem arrays
	float h_1d_in[ARRAY_SIZE];
	float h_1d_out[ARRAY_SIZE];
	printf("Original Array: \n");
	for(int i = 0; i < ARRAY_SIZE;i++)
	{
		h_1d_in[i] = float(i);
		printf("%d " , i);
	}
	printf("\n");

	//Device mem arrays
	float * d_1d_in;
	float * d_1d_out;
	
	hipMalloc((void **) &d_1d_in,ARRAY_BYTES);
	hipMalloc((void **) &d_1d_out, ARRAY_BYTES);
	
	hipMemcpy(d_1d_in,h_1d_in,ARRAY_BYTES,hipMemcpyHostToDevice);
	
	square_1d_vector<<< 1, ARRAY_SIZE >>>(d_1d_out , d_1d_in);
	
	hipMemcpy(h_1d_out,d_1d_out,ARRAY_BYTES,hipMemcpyDeviceToHost);

	printf("Square Array : \n");
	for(int i = 0;i < ARRAY_SIZE; i++)
	{
		printf("%d ",int(h_1d_out[i]));
	}
	printf("\n");
	
	hipFree(d_1d_in);
	hipFree(d_1d_out);
}

void print_header(void)
{
	printf(" ============================================== \n");
	printf(" ===== PARALLEL PROGRAMMING (1D - VECTOR) ===== \n");
	printf(" ============================================== \n");
	printf("\n");
}

int main(int argc,char** argv)
{
	print_header();
	printf("Computing square of numbers in some array ... \n");	
	call_1d_parallel_computing();
	return 0;
}
