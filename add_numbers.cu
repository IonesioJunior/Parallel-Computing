
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void add_2d_numbers(int *d_out,int *d_in)
{
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int index = row * col + row;
	if(index == 8){
		printf("Checkpoint!\n");	
	}
	d_out[index] = d_in[index];
}


void call_2d_parallel_computing(void)
{
	const int N_ROWS = 5;
	const int N_COLS = 5;
	const int BYTES_SIZE = N_ROWS * N_COLS * sizeof(int);

	// Define Host matrix
	int h_2d_in[N_ROWS][N_COLS];
	int h_2d_out[N_ROWS][N_COLS];
	for(int i = 0; i < N_ROWS;i++)
	{
		for(int j = 0; j < N_COLS;j++)
		{
			h_2d_in[i][j] = i + j;
			printf("%d ",i + j);
		}
		printf("\n");
	}
	printf("\n");

	// Define device matrix
	int * d_2d_in;
	int * d_2d_out;

	hipMalloc((void **) &d_2d_in,BYTES_SIZE);
	hipMalloc((void **) &d_2d_out,BYTES_SIZE);

	hipMemcpy(d_2d_in,h_2d_in,BYTES_SIZE,hipMemcpyHostToDevice);

	dim3 dimBlock(N_COLS,N_ROWS);
	dim3 dimGrid(1,1);

	add_2d_numbers<<<dimGrid,dimBlock>>>(d_2d_out,d_2d_in);

	hipMemcpy(h_2d_out,d_2d_out,BYTES_SIZE,hipMemcpyDeviceToHost);	

	printf("Result : \n" );
	for(int i = 0 ; i < N_ROWS;i++){
		for(int j = 0 ; j < N_COLS;j++){
			printf("%d ",h_2d_out[i][j]);
		}
		printf("\n");
	}
	printf("\n");

	hipFree(d_2d_in);
	hipFree(d_2d_out);
}


int main(int argc,char ** argv)
{
	call_2d_parallel_computing();
	return 0;
}
