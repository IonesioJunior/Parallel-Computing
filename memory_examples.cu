
#include <hip/hip_runtime.h>
#include <stdio.h>

#define ARRAY_SIZE 128
__global__ void avarage_list(float * value_in,float * value_out){
	//Local memory
	int index = threadIdx.x;
	float sum = 0.0;

	// Static shared var
	__shared__ float sh_arr[ARRAY_SIZE];
	
	//Shared mem  | global memory 
	sh_arr[index] = value_in[index];
	
	__syncthreads(); // Garante que todos os numeros foram copiados antes de começar a prox operaçao
	
	//shared memory operation
	for(int i = 0; i <= index;i++){
		sum += sh_arr[i];
	}
	
	// Global memory | local memory
	value_out[index] = sum / (index + 1);
}



int main(int argc,char** argv){
	const int BYTE_SIZE = ARRAY_SIZE * sizeof(float);
	
	//Host var
	float h_values_in[ARRAY_SIZE];
	float h_avarage_out[ARRAY_SIZE];
	
	printf("Array Values : \n");
	for(int i = 0 ; i < ARRAY_SIZE;i++){
		h_values_in[i] = float(i * 2);
		printf("%.2f " , h_values_in[i]);
	}
	printf("\n");
	
	
	//Device var
	float *d_values_in;
	float *d_avarage_out;
	hipMalloc((void**) &d_values_in,BYTE_SIZE);
	hipMemcpy(d_values_in,h_values_in,BYTE_SIZE,hipMemcpyHostToDevice);
	hipMalloc((void**) &d_avarage_out,BYTE_SIZE);
	
	avarage_list<<<1,ARRAY_SIZE>>>(d_values_in,d_avarage_out);
	
	hipMemcpy(h_avarage_out,d_avarage_out,BYTE_SIZE,hipMemcpyDeviceToHost);
		

	printf("Avarage Array: \n");	
	for(int i = 0 ; i < ARRAY_SIZE;i++){
		printf("%.2f ",h_avarage_out[i]);
	}
	printf("\n");

	hipFree(d_values_in);
	hipFree(d_avarage_out);
	return 0;
}
