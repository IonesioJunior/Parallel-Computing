
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void shift_forward(int * value)
{
	int index = threadIdx.x;
	__shared__ int array[64];
	array[index] = threadIdx.x; 
	__syncthreads(); // Garantir que todos os valores foram armazenados antes de começar o shift
	if(index < 63)
	{
		int tmp = array[index + 1];
		__syncthreads(); // Salvar cada valor antes que o mesmo seja trocado por outra thread
		value[index] = tmp;
		__syncthreads();
	}
} 


int main(int argc,char ** argv)
{
	
	const int ARRAY_SIZE = 64;
	const int SIZE = ARRAY_SIZE  * sizeof(int);
	
	int * d_out;
	hipMalloc((void **) &d_out,SIZE);
	
	shift_forward<<<1,64>>>(d_out);	
	
	int h_out[ARRAY_SIZE];
	hipMemcpy(h_out,d_out,SIZE,hipMemcpyDeviceToHost);

	for(int i = 0 ; i < ARRAY_SIZE;i++)
	{
		printf("%d ",h_out[i]);
	}
	printf("\n");
	return 0;
}
